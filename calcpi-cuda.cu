#include <cstdio>
#include <hip/hip_runtime.h>

double do_calcpi(int iterations) {
    hipDeviceProp_t deviceProp = {};
    hipGetDeviceProperties(&deviceProp, 0);
    printf("%d\n", deviceProp.maxThreadsPerBlock);
    return 3.14;
}

extern "C" {

double calc_pi(int iterations) {

    return do_calcpi(iterations);
}

};

