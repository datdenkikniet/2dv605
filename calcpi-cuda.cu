#include "hip/hip_runtime.h"
extern "C" {

#include <hip/hip_runtime.h>
#include <cstdio>
#include "timer.h"

__global__ void pi_iter(const int *offset, const int *iterations, const double *m, double *pieparts) {
    unsigned int index = blockIdx.x * blockDim.x + threadIdx.x + *offset;
    if (index < *iterations) {
        double n_i = ((double) index + 0.5) * *m;
        pieparts[index - *offset] = 4.0 / (1.0 + n_i * n_i);
    }
}

// Add extra timers to determine actual computation time (and exclude malloc time)
double do_calcpi(int worksize, int iterations) {
    int BATCH_SIZE = worksize;
    int *device_offset;
    hipMalloc(&device_offset, sizeof(int));

    int *device_iterations;
    hipMalloc(&device_iterations, sizeof(int));
    hipMemcpy(&iterations, device_iterations, sizeof(int), hipMemcpyHostToDevice);

    double *device_pieparts;
    hipMalloc(&device_pieparts, sizeof(double) * BATCH_SIZE);

    double *host_pieparts = (double *) malloc(sizeof(double) * BATCH_SIZE);

    double m = 1.0 / (double) iterations;

    double *device_m;
    hipMalloc(&device_m, sizeof(double));
    hipMemcpy(device_m, &m, sizeof(double), hipMemcpyHostToDevice);

    double mypi = 0.0;
    for (int i = 0; i < iterations; i += BATCH_SIZE) {
        int actualSize = BATCH_SIZE;
        if (actualSize > (iterations - i)) {
            actualSize = iterations - i;
        }
        hipMemcpy(device_offset, &i, sizeof(int), hipMemcpyHostToDevice);
        int blocks = (BATCH_SIZE / 1024) + 1;
        pi_iter<<<blocks, 1024>>>(device_offset, device_iterations, device_m, device_pieparts);

        hipMemcpy(host_pieparts, device_pieparts, sizeof(double) * BATCH_SIZE, hipMemcpyDeviceToHost);
        for (int k = 0; k < actualSize; k++) {
            mypi += host_pieparts[k];
        }
    }
    mypi *= m;
    hipFree(device_pieparts);
    free(host_pieparts);

    return mypi;
}
double calc_pi(int worksize, int iterations) {
    return do_calcpi(worksize, iterations);
}

};